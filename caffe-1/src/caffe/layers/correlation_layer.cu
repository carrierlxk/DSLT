#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#include "caffe/caffe.hpp"

#define ROUND_OFF 50000

#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

namespace caffe {

// == Dimension rearrangement Kernel
  
template <typename Dtype>
__global__ void blob_rearrange_kernel2(const Dtype* in, Dtype* out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    int xy = blockIdx.x*blockDim.x + threadIdx.x;
    if(xy>=widthheight)
        return;

    int ch = blockIdx.y;
    int n  = blockIdx.z;

    Dtype value=in[(n*channels+ch)*widthheight+xy];

    __syncthreads();

    int xpad  = (xy % width + padding);
    int ypad  = (xy / width + padding);
    int xypad = ypad * (width+2*padding) + xpad;

    out[(n*pwidthheight+xypad)*channels + ch] = value;
}

// == Correlation Kernel
template <typename Dtype> 
__global__ void CorrelateData(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top) 
{
  extern __shared__ char patch_data_char[];
  
  Dtype *patch_data = (Dtype *)patch_data_char;
  
    // First (upper left) position of kernel upper-left corner in current center position of neighborhood in image 1
  int x1 = blockIdx.x*stride1 + max_displacement;
  int y1 = blockIdx.y*stride1 + max_displacement;
  int item = blockIdx.z;
  int ch_off = threadIdx.x;
  
  // Load 3D patch into shared shared memory
  for(int j = 0; j < kernel_size; j++) { // HEIGHT
    for(int i = 0; i < kernel_size; i++) { // WIDTH
      int ji_off = ((j * kernel_size) + i) * bottomchannels;
      for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
          int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
          int idxPatchData = ji_off + ch;
          patch_data[idxPatchData] = bottom0[idx1];
      }
    }
  }
  
  __syncthreads();
  
  __shared__ Dtype sum[WARPS_PER_BLOCK*THREADS_PER_WARP];
  
  // Compute correlation
  for(int top_channel = 0; top_channel < topchannels; top_channel++) {
    sum[ch_off] = 0;
  
    int s2o = (top_channel % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    int s2p = (top_channel / neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    
    for(int j = 0; j < kernel_size; j++) { // HEIGHT
      for(int i = 0; i < kernel_size; i++) { // WIDTH
        int ji_off = ((j * kernel_size) + i) * bottomchannels;
        for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
          int x2 = x1 + s2o;
          int y2 = y1 + s2p;
          
          int idxPatchData = ji_off + ch;
          int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + ch;
          
          sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
        }
      }
    }
    
    __syncthreads();
    
    if(ch_off == 0) {
        Dtype total_sum = 0;
        for(int idx = 0; idx < WARPS_PER_BLOCK*THREADS_PER_WARP; idx++) {
            total_sum += sum[idx];
        }
        const int sumelems = kernel_size*kernel_size*bottomchannels;
        const int index = ((top_channel*topheight + blockIdx.y)*topwidth)+blockIdx.x;
        top[index + item*topcount] = total_sum / (float)sumelems;
    }
  }
  
  
  // Aggregate  
}

// == Correlation Backward Pass Kernel (For Blob 0)
template <typename Dtype> 
__global__ void CorrelateDataBackward0(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom1, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int n = index % bottomchannels; //channels
    int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size; //h-pos

    //Get X,Y ranges and clamp
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    
    // Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off; // floor (l - max_displacement) / stride1
    int ymax = (m - max_displacement + round_off_s1) / stride1 - round_off; // floor (m - max_displacement) / stride1
    

    Dtype sum = 0;
    if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
    {
        xmin = max(0,xmin);
        xmax = min(topwidth-1,xmax);

        ymin = max(0,ymin);
        ymax = min(topheight-1,ymax);

        for(int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
          for(int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {

            // Get bottom1 data:
            int s2o = stride2 * o;
            int s2p = stride2 * p;
            int idxbot1 = ((item * pbottomheight + (m+s2p)) * pbottomwidth + (l+s2o)) * bottomchannels + n;
            Dtype bot1tmp = bottom1[idxbot1]; // bottom1[l+s2o,m+s2p,n]

            // Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width + (o+neighborhood_grid_radius); // index [o,p]
            int idxopoffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * bot1tmp;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
		const int bot0index = ((n * bottomheight) + (m-pad_size)) * bottomwidth + (l-pad_size);
    bottom0diff[bot0index + item*bottomcount] = sum / (float)sumelems;
  }

}



// == Correlation Backward Pass Kernel (For Blob 1)
template <typename Dtype> 
__global__ void CorrelateDataBackward1(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  const Dtype *bottom0, Dtype *bottom1diff, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    //int l = index % bottomwidth + pad_size; //w-pos
    //int m = (index / bottomwidth) % bottomheight + pad_size; //h-pos
    //int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels
    int n = index % bottomchannels; //channels
    int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size; //h-pos
    
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    Dtype sum = 0;
    for(int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
      for(int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {
        
        int s2o = stride2 * o;
        int s2p = stride2 * p;
        
        //Get X,Y ranges and clamp
        // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - max_displacement - s2p + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        
        // Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off; // floor (l - max_displacement - s2o) / stride1
        int ymax = (m - max_displacement - s2p + round_off_s1) / stride1 - round_off; // floor (m - max_displacement - s2p) / stride1

        if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
        {
            xmin = max(0,xmin);
            xmax = min(topwidth-1,xmax);

            ymin = max(0,ymin);
            ymax = min(topheight-1,ymax);

            // Get bottom0 data:
            int idxbot0 = ((item * pbottomheight + (m-s2p)) * pbottomwidth + (l-s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot0]; // bottom1[l+s2o,m+s2p,n]

            // Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width + (o+neighborhood_grid_radius); // index [o,p]
            int idxOpOffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * bot0tmp;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
		const int bot1index = ((n * bottomheight) + (m-pad_size)) * bottomwidth + (l-pad_size);
		bottom1diff[bot1index + item*bottomcount] = sum / (float)sumelems;
  }

}

// == Correlation Kernel Subtraction
template <typename Dtype> 
__global__ void CorrelateDataSubtract(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int topcount,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int x = index % topwidth; //w-pos
    int y = (index / topwidth) % topheight; //h-pos
    int c = (index / topwidth / topheight) % topchannels; //channels
        
    // Offset of patch in image 2
    int s2o = (c % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    int s2p = (c / neighborhood_grid_width - neighborhood_grid_radius) * stride2;
        
    // First (upper left) position of kernel center in current neighborhood in image 1
    int x1 = x*stride1 + kernel_radius + max_displacement;
    int y1 = y*stride1 + kernel_radius + max_displacement;
    
    // Iterate through 3D patch
    Dtype sum = 0;
    for(int j = -kernel_radius; j <= kernel_radius; j++) { // HEIGHT
      for(int i = -kernel_radius; i <= kernel_radius; i++) { // WIDTH
        for(int l = 0; l < bottomchannels; l++) { // CHANNELS
          // Calculate position in image 2
          int x2 = x1 + s2o;
          int y2 = y1 + s2p;

          // Indices in bottom data: (CH=l,W=x2,H=y2,N)
          int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + l;
          int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + l;

          // Do the correlation:
          sum += fabsf(bottom0[idx1] - bottom1[idx2]);
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    top[index + item*topcount] = sum / (float)sumelems;
  }

}


// == Correlation Backward Pass Kernel (For Blob 0)
template <typename Dtype> 
__global__ void CorrelateDataBackward0Subtract(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom0, const Dtype *bottom1, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int l = index % bottomwidth + pad_size; //w-pos
    int m = (index / bottomwidth) % bottomheight + pad_size; //h-pos
    int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels

    //Get X,Y ranges and clamp
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    
    // Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off; // floor (l - max_displacement) / stride1
    int ymax = (m - max_displacement + round_off_s1) / stride1 - round_off; // floor (m - max_displacement) / stride1
    

    Dtype sum = 0;
    if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
    {
        xmin = max(0,xmin);
        xmax = min(topwidth-1,xmax);

        ymin = max(0,ymin);
        ymax = min(topheight-1,ymax);

        for(int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
          for(int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {

            // Get bottom1 data:
            int s2o = stride2 * o;
            int s2p = stride2 * p;
            int idxbot = ((item * pbottomheight + (m+s2p)) * pbottomwidth + (l+s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot]; // bottom0[l+s2o,m+s2p,n]
            Dtype bot1tmp = bottom1[idxbot]; // bottom1[l+s2o,m+s2p,n]
            Dtype sign = (bot0tmp >= bot1tmp) ? Dtype(1.0) : Dtype(-1.0);

            // Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width + (o+neighborhood_grid_radius); // index [o,p]
            int idxopoffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * sign;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    bottom0diff[index + item*bottomcount] = sum / (float)sumelems;
  }

}


// == Correlation Backward Pass Kernel (For Blob 1)
template <typename Dtype> 
__global__ void CorrelateDataBackward1Subtract(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *bottom1diff, const Dtype *topdiff) 
{
  CUDA_KERNEL_LOOP(index, nthreads) {
    int l = index % bottomwidth + pad_size; //w-pos
    int m = (index / bottomwidth) % bottomheight + pad_size; //h-pos
    int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels
    
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;
    
    Dtype sum = 0;
    for(int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
      for(int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {
        
        int s2o = stride2 * o;
        int s2p = stride2 * p;
        
        //Get X,Y ranges and clamp
        // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - max_displacement - s2p + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        
        // Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off; // floor (l - max_displacement - s2o) / stride1
        int ymax = (m - max_displacement - s2p + round_off_s1) / stride1 - round_off; // floor (m - max_displacement - s2p) / stride1

        if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
        {
            xmin = max(0,xmin);
            xmax = min(topwidth-1,xmax);

            ymin = max(0,ymin);
            ymax = min(topheight-1,ymax);

            // Get bottom0 data:
            int idxbot = ((item * pbottomheight + (m-s2p)) * pbottomwidth + (l-s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot]; // bottom0[l+s2o,m+s2p,n]
            Dtype bot1tmp = bottom1[idxbot]; // bottom1[l+s2o,m+s2p,n]
            Dtype sign = (bot0tmp >= bot1tmp) ? Dtype(-1.0) : Dtype(1.0);

            // Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width + (o+neighborhood_grid_radius); // index [o,p]
            int idxOpOffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                sum += topdiff[idxtopdiff] * sign;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    bottom1diff[index + item*bottomcount] = sum / (float)sumelems;
  }

}
// == Forward 

template <typename Dtype>
void CorrelationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top)
{
    CHECK_EQ(bottom.size(),2);
    CHECK_EQ(top.size(),1);

    const int bnum = bottom[0]->num();
    const int bchannels = bottom[0]->channels();
    const int bheight = bottom[0]->height();
    const int bwidth = bottom[0]->width();
    const int bwidthheight = bwidth * bheight;

    const int topcount = top_width_ * top_height_ * top_channels_;
    
    dim3 threadsPerBlock(THREADS_PER_WARP * WARPS_PER_BLOCK); 
    
    hipMemset(rbot1_->mutable_gpu_data(), 0, rbot1_->count()*sizeof(Dtype));
    hipMemset(rbot2_->mutable_gpu_data(), 0, rbot2_->count()*sizeof(Dtype));
    
    int threads_per_block=16;
    dim3 totalBlocksRearr((bwidthheight-1)/threads_per_block+1, bchannels, bnum);
    const int pwidthheight = (bwidth + 2 * pad_size_) * (bheight + 2 * pad_size_);
    
    blob_rearrange_kernel2<Dtype><<<totalBlocksRearr,threads_per_block>>>
            (bottom[0]->gpu_data(),rbot1_->mutable_gpu_data(),bnum,bchannels,bwidth,bheight,bwidthheight,pad_size_,pwidthheight);
    
    blob_rearrange_kernel2<Dtype><<<totalBlocksRearr,threads_per_block>>>
            (bottom[1]->gpu_data(),rbot2_->mutable_gpu_data(),bnum,bchannels,bwidth,bheight,bwidthheight,pad_size_,pwidthheight);
    
    const int num = bnum;
    const int channels = bchannels;
    const int height = bheight + 2*pad_size_;
    const int width = bwidth + 2*pad_size_;
    
    const int shared_memory_per_block = (kernel_size_*kernel_size_)*bchannels;

    if(corr_type_ == CorrelationParameter_CorrelationType_MULTIPLY) {
        // CorrelationLayer
        int topThreadCount = topcount;
        
        dim3 totalBlocksCorr(top_width_, top_height_, num);
        
        
        CorrelateData<Dtype><<<totalBlocksCorr, threadsPerBlock, shared_memory_per_block * sizeof(Dtype)>>>(
            topThreadCount,
            num, top_width_, top_height_, top_channels_, topcount,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_, kernel_size_,
            stride1_, stride2_,
            width, height, channels,
            rbot1_->gpu_data(), rbot2_->gpu_data(), top[0]->mutable_gpu_data()
            );

        CUDA_POST_KERNEL_CHECK;
        
    } else if(corr_type_ == CorrelationParameter_CorrelationType_SUBTRACT) {
        // CorrelationLayer
        for(int n = 0; n < num; n++) {
            
            int topThreadCount = topcount;
            
            CorrelateDataSubtract<Dtype><<<CAFFE_GET_BLOCKS(topThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
                topThreadCount,
                num, n, top_width_, top_height_, top_channels_, topcount,
                max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
                stride1_, stride2_,
                width, height, channels,
                rbot1_->gpu_data(), rbot2_->gpu_data(), top[0]->mutable_gpu_data()
                );

            
            CUDA_POST_KERNEL_CHECK;
        }
    }
}


template <typename Dtype>
void CorrelationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{

    // Get top diff, compute bottom diff
    const Dtype* top_diff = top[0]->gpu_diff();

    Dtype* bottom0_diff = bottom[0]->mutable_gpu_diff();
    Dtype* bottom1_diff = bottom[1]->mutable_gpu_diff();

    const Dtype* bottom0_data = bottom[0]->gpu_data();
    const Dtype* bottom1_data = bottom[1]->gpu_data();

    const int num = bottom[0]->num();
    const int channels = bottom[0]->channels();
    const int height = bottom[0]->height();
    const int width = bottom[0]->width();

    const int paddedheight = height + 2*pad_size_;
    const int paddedwidth = width + 2*pad_size_;

    const int bottomcount = channels * height * width;

    int botThreadCount = bottomcount;
   
    // CorrelationLayerBackward
    
    bottom0_diff = bottom[0]->mutable_gpu_diff();
    bottom1_diff = bottom[1]->mutable_gpu_diff();

    if(corr_type_ == CorrelationParameter_CorrelationType_MULTIPLY) {
        
        // == Run kernel Backward 0
        dim3 totalBlocksBackward0(width, height, channels * num); //First dim is fastest
        dim3 threadsPerBlockBackward0(THREADS_PER_WARP * WARPS_PER_BLOCK); 
        const int buffer_size_backw0 = ((int)ceil((float)(2 * kernel_radius_) / (float)stride1_) + 1) * top_channels_;
       
        // == Run kernel Backward 0 
        for(int n = 0; n < num; n++) {
        //Bottom0:
        CorrelateDataBackward0<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            bottom0_diff, rbot2_->gpu_data(), top_diff
            ); 
    
        CUDA_POST_KERNEL_CHECK;
        }
        
        // == Run kernel Backward 1
        for(int n = 0; n < num; n++) {
        CorrelateDataBackward1<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            rbot1_->gpu_data(), bottom1_diff, top_diff
            );
    
        CUDA_POST_KERNEL_CHECK;
        }
        
    } else if(corr_type_ == CorrelationParameter_CorrelationType_SUBTRACT) {
        for(int n = 0; n < num; n++) {
        //Bottom0:
        CorrelateDataBackward0Subtract<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            bottom0_diff, rbot1_->gpu_data(), rbot2_->gpu_data(), top_diff
            );
    
        CUDA_POST_KERNEL_CHECK;
        }

        for(int n = 0; n < num; n++) {
        //Bottom0:
        CorrelateDataBackward1Subtract<Dtype><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
            botThreadCount,
            num, n, top_width_, top_height_, top_channels_,
            max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
            stride1_, stride2_,
            width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
            rbot1_->gpu_data(), rbot2_->gpu_data(), bottom1_diff, top_diff
            );
    
        CUDA_POST_KERNEL_CHECK;
        }
    }
}


INSTANTIATE_LAYER_GPU_FUNCS(CorrelationLayer);

}  // namespace caffe
